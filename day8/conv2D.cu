
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define EPSILON 1e-3

__global__ void conv2D(float *input, float *output, float *filter, int radius, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) 
    {
        float val = 0.0f;
        int f_size = 2 * radius + 1;

        for(int row_f = 0; row_f < f_size; row_f++) 
        {
            for(int col_f = 0; col_f < f_size; col_f++) 
            {
                int cur_row = row - radius + row_f;
                int cur_col = col - radius + col_f;

                if(cur_row >=0 && cur_row < N && cur_col >= 0 && cur_col < N) 
                {
                    val += filter[row_f * f_size + col_f] * input[cur_row * N + cur_col]; 
                }
            }
        }
        output[row * N + col] = val;
    }
}

void conv2D_cpu(float *input, float *output, float *filter, int radius, int N) 
{
    int f_size = 2 * radius + 1;

    for(int row = 0; row < N; row++) 
    {
        for(int col = 0; col < N; col++) 
        {
            float val = 0.0f;

            for(int row_f = 0; row_f < f_size; row_f++) 
            {
                for(int col_f = 0; col_f < f_size; col_f++) 
                {
                    int cur_row = row - radius + row_f;
                    int cur_col = col - radius + col_f;
                    
                    if(cur_row >= 0 && cur_row < N && cur_col >= 0 && cur_col < N) 
                    {
                        val += filter[row_f * f_size + col_f] * input[cur_row * N + cur_col];
                    }
                }
            }
            output[row * N + col] = val;
        }
    }
}

__global__ void heNormal(float *filter, int radius, unsigned int seed) {
    int t_x = blockIdx.x * blockDim.x + threadIdx.x;
    int f_size = 2 * radius + 1;
    int total = f_size * f_size;

    if(t_x < total) 
    {
        float fan_in = (float)total;
        float std_dev = sqrtf(2.0f / fan_in);
        
        hiprandState state;
        hiprand_init(seed, t_x, 0, &state);
        filter[t_x] = hiprand_normal(&state) * std_dev;
    }
}

__global__ void verify(float *gpu_res, float *cpu_res, int *errors, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < N && col < N) 
    {
        int idx = row * N + col;
        if(fabsf(gpu_res[idx] - cpu_res[idx]) > EPSILON) 
        {
            atomicAdd(errors, 1);
        }
    }
}

__global__ void init(float *input, int N, unsigned int seed) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) 
    {
        hiprandState state;
        hiprand_init(seed, row * N + col, 0, &state);
        input[row * N + col] = hiprand_uniform(&state);
    }
}

int main() {
    int N = 128;
    int radius = 1;
    float gpu_time, cpu_time;

    float *in_gpu, *out_gpu, *filter, *cpu_res_gpu;
    int *d_errors;

    hipMalloc(&in_gpu, N*N*sizeof(float));
    hipMalloc(&out_gpu, N*N*sizeof(float));
    hipMalloc(&filter, (2*radius+1)*(2*radius+1)*sizeof(float));
    hipMalloc(&cpu_res_gpu, N*N*sizeof(float));
    hipMalloc(&d_errors, sizeof(int));

    dim3 init_threads(16, 16);
    dim3 init_blocks((N+15)/16, (N+15)/16);
    init<<<init_blocks, init_threads>>>(in_gpu, N, time(NULL));
    
    size_t f_size = (2*radius+1)*(2*radius+1);
    heNormal<<<(f_size+255)/256, 256>>>(filter, radius, time(NULL));
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    dim3 conv_threads(16, 16);
    dim3 conv_blocks((N+15)/16, (N+15)/16);
    
    hipEventRecord(start);
    conv2D<<<conv_blocks, conv_threads>>>(in_gpu, out_gpu, filter, radius, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    float *input_cpu = (float*)malloc(N*N*sizeof(float));
    float *filter_cpu = (float*)malloc((2*radius+1)*(2*radius+1)*sizeof(float));
    float *output_cpu = (float*)malloc(N*N*sizeof(float));
    
    hipMemcpy(input_cpu, in_gpu, N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(filter_cpu, filter, (2*radius+1)*(2*radius+1)*sizeof(float), hipMemcpyDeviceToHost);

    clock_t cpu_start = clock();
    conv2D_cpu(input_cpu, output_cpu, filter_cpu, radius, N);
    cpu_time = ((double)(clock() - cpu_start) / CLOCKS_PER_SEC) * 1000;

    hipMemcpy(cpu_res_gpu, output_cpu, N*N*sizeof(float), hipMemcpyHostToDevice);

    hipMemset(d_errors, 0, sizeof(int));
    verify<<<conv_blocks, conv_threads>>>(out_gpu, cpu_res_gpu, d_errors, N);
    
    int errors;
    hipMemcpy(&errors, d_errors, sizeof(int), hipMemcpyDeviceToHost);
    printf("Mismatches: %d\n", errors);
    printf("GPU Convolution Execution Time: %.2f ms\n", gpu_time);
    printf("CPU Convolution Execution Time: %.2f ms\n", cpu_time);

    free(input_cpu);
    free(filter_cpu);
    free(output_cpu);
    hipFree(in_gpu);
    hipFree(out_gpu);
    hipFree(filter);
    hipFree(cpu_res_gpu);
    hipFree(d_errors);

    return 0;
}