
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hipsparse.h>

#define N (1<<12)
#define THREADS 1024
#define BLOCKS ((N * N + THREADS - 1) / THREADS)
#define SPARSITY 0.1f

__global__ void init(int *sparse_m, unsigned long seed) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = N * N;
    if (tid < totalElements) 
    {
        hiprandState state;
        hiprand_init(seed, tid, 0, &state);
        float rand_val = hiprand_uniform(&state);
        sparse_m[tid] = (rand_val <= SPARSITY) ? (hiprand(&state) % 100 + 1) : 0;
    }
}

void COO(int *sparse_m, int **cooRow, int **cooCol, int **cooVal, int64_t *non_zero) 
{
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseDnMatDescr_t matrix;
    hipsparseCreateDnMat(&matrix, N, N, N, sparse_m, HIP_R_32I, HIPSPARSE_ORDER_ROW);

    *non_zero = 0;
    for (int i = 0; i < N * N; i++) 
    {
        if (sparse_m[i] != 0) (*non_zero)++;
    }

    hipMallocManaged(cooRow, (*non_zero) * sizeof(int));
    hipMallocManaged(cooCol, (*non_zero) * sizeof(int));
    hipMallocManaged(cooVal, (*non_zero) * sizeof(int));

    hipsparseSpMatDescr_t matCOO;
    hipsparseCreateCoo(&matCOO, N, N, *non_zero, *cooRow, *cooCol, *cooVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I);

    size_t bufferSize = 0;
    void *dBuffer = NULL;
    hipsparseDenseToSparseAlg_t alg = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
    
    hipsparseDenseToSparse_bufferSize(handle, matrix, matCOO, alg, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseDenseToSparse_analysis(handle, matrix, matCOO, alg, &bufferSize);
    hipsparseDenseToSparse_convert(handle, matrix, matCOO, alg, dBuffer);

    hipFree(dBuffer);
    hipsparseDestroyDnMat(matrix);
    hipsparseDestroySpMat(matCOO);
    hipsparseDestroy(handle);
}

void CSR(int *sparse_m, int **csrRowPtr, int **csrCol, int **csrVal, int64_t *non_zero) 
{
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseDnMatDescr_t matrix;
    hipsparseCreateDnMat(&matrix, N, N, N, sparse_m, HIP_R_32I, HIPSPARSE_ORDER_ROW);

    *non_zero = 0;
    for (int i = 0; i < N * N; i++) 
    {
        if (sparse_m[i] != 0) (*non_zero)++;
    }

    hipMallocManaged(csrRowPtr, (N + 1) * sizeof(int));
    hipMallocManaged(csrCol, (*non_zero) * sizeof(int));
    hipMallocManaged(csrVal, (*non_zero) * sizeof(int));

    hipsparseSpMatDescr_t matCSR;
    hipsparseCreateCsr(&matCSR, N, N, *non_zero, *csrRowPtr, *csrCol, *csrVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I);

    size_t bufferSize = 0;
    void *dBuffer = NULL;
    hipsparseDenseToSparseAlg_t alg = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
    
    hipsparseDenseToSparse_bufferSize(handle, matrix, matCSR, alg, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseDenseToSparse_analysis(handle, matrix, matCSR, alg, &bufferSize);
    hipsparseDenseToSparse_convert(handle, matrix, matCSR, alg, dBuffer);

    hipFree(dBuffer);
    hipsparseDestroyDnMat(matrix);
    hipsparseDestroySpMat(matCSR);
    hipsparseDestroy(handle);
}

int main() 
{
    int *sparse_m, *cooRow, *cooCol, *cooVal, *csrRowPtr, *csrCol, *csrVal;
    int64_t non_zero_coo = 0, non_zero_csr = 0;

    hipMallocManaged(&sparse_m, N * N * sizeof(int));

    init<<<BLOCKS, THREADS>>>(sparse_m, time(NULL));
    hipDeviceSynchronize();

    COO(sparse_m, &cooRow, &cooCol, &cooVal, &non_zero_coo);

    CSR(sparse_m, &csrRowPtr, &csrCol, &csrVal, &non_zero_csr);

    printf("Dense Matrix Storage Size: %.2lf MB\n", (N * N * sizeof(int) / (1024.0 * 1024)));
    printf("Sparse Matrix Storage Size (COO): %.2lf MB\n", ((non_zero_coo * 3 * sizeof(int)) / (1024.0 * 1024)));
    printf("Sparse Matrix Storage Size (CSR): %.2lf MB\n", (((N + 1) * sizeof(int) + non_zero_csr * 2 * sizeof(int)) / (1024.0 * 1024)));

    hipFree(sparse_m);
    hipFree(cooRow);
    hipFree(cooCol);
    hipFree(cooVal);
    hipFree(csrRowPtr);
    hipFree(csrCol);
    hipFree(csrVal);

    return 0;
}
