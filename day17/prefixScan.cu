
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define N (1<<26)
#define THREADS 1024
#define BLOCKS ((N + THREADS - 1) / THREADS)

void koggeCPU(int *x, int *y) 
{
    y[0] = x[0];
    for (int i = 1; i < N; i++) 
    {
        y[i] = y[i - 1] + x[i];
    }
}

__global__ void koggeGPU(int *x, int *y, int *partialSums) 
{
    int t_i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int x_s[THREADS];

    if (t_i < N)
    {
        x_s[threadIdx.x] = x[t_i];
    }
    else
    {
        x_s[threadIdx.x] = 0;
    }
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) 
    {
        __syncthreads();
        int temp = 0;
        if (threadIdx.x >= stride) 
        {
            temp = x_s[threadIdx.x] + x_s[threadIdx.x - stride];
        }
        __syncthreads();
        if (threadIdx.x >= stride) 
        {
            x_s[threadIdx.x] = temp;
        }
    }

    if (threadIdx.x == blockDim.x - 1)
    {
        partialSums[blockIdx.x] = x_s[threadIdx.x];
    }

    if (t_i < N)
    {
        y[t_i] = x_s[threadIdx.x];
    }

}

__global__ void koggeDoubleBufferGPU(int *x, int *y, int *partialSums) 
{
    int t_i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int shared1[THREADS];
    __shared__ int shared2[THREADS];

    int *inShared = shared1;
    int *outShared = shared2;

    if (t_i < N)
    {
        shared1[threadIdx.x] = x[t_i];
    }
    else
    {
        shared1[threadIdx.x] = 0;
    }
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) 
    {
        if (threadIdx.x >= stride) 
        {
            outShared[threadIdx.x] = inShared[threadIdx.x] + inShared[threadIdx.x - stride];
        }
        else
        {
            outShared[threadIdx.x] = inShared[threadIdx.x];
        }
        __syncthreads();

        int *t = inShared;
        inShared = outShared;
        outShared = t;
    }

    if (threadIdx.x == blockDim.x - 1)
    {
        partialSums[blockIdx.x] = inShared[threadIdx.x];
    }

    if (t_i < N)
    {
        y[t_i] = inShared[threadIdx.x];
    }
}

__global__ void add(int *y, int *partialSums) 
{
    int t_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (blockIdx.x > 0 && t_i < N)
    {
        y[t_i] += partialSums[blockIdx.x - 1];
    }
}

int main() 
{
    int *input, *output_cpu, *output_gpu, *output_gpu_double_shared, *partialSums1, *partialSums2;
    size_t size = N * sizeof(int);

    hipMallocManaged(&input, size);
    output_cpu = (int*)malloc(size);
    hipMallocManaged(&output_gpu, size);
    hipMallocManaged(&output_gpu_double_shared, size);
    hipMallocManaged(&partialSums1, BLOCKS * sizeof(int));
    hipMallocManaged(&partialSums2, BLOCKS * sizeof(int));

    for (int i = 0; i < N; i++) 
    {
        input[i] = 1;
    }
    hipMemset(output_gpu, 0, size);
    hipMemset(output_gpu_double_shared, 0, size);
    hipMemset(partialSums1, 0, BLOCKS * sizeof(int));
    hipMemset(partialSums2, 0, BLOCKS * sizeof(int));

    float cpu_time, gpu_time, gpu_time_double_shared;
    clock_t cpu_start = clock();
    koggeCPU(input, output_cpu);
    cpu_time = ((double)(clock() - cpu_start) / CLOCKS_PER_SEC) * 1000;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    koggeGPU<<<BLOCKS, THREADS>>>(input, output_gpu, partialSums1);
    hipDeviceSynchronize();

    for (int i = 1; i < BLOCKS; i++) 
    {
        partialSums1[i] += partialSums1[i - 1];
    }

    add<<<BLOCKS, THREADS>>>(output_gpu, partialSums1);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    koggeDoubleBufferGPU<<<BLOCKS, THREADS>>>(input, output_gpu_double_shared, partialSums2);
    hipDeviceSynchronize();

    for (int i = 1; i < BLOCKS; i++) 
    {
        partialSums2[i] += partialSums2[i - 1];
    }

    add<<<BLOCKS, THREADS>>>(output_gpu_double_shared, partialSums2);
    hipDeviceSynchronize();

    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&gpu_time_double_shared, start2, stop2);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    if (output_cpu[N - 1] != output_gpu[N - 1] || output_gpu[N - 1] != output_gpu_double_shared[N - 1])
    {
        printf("Failure\n");
    }
    else
    {
        printf("Success\n");
    }

    printf("\nCPU execution time: %.4f ms\n", cpu_time);
    printf("GPU execution time: %.4f ms\n", gpu_time);
    printf("GPU execution time (Double Buffer): %.4f ms\n", gpu_time_double_shared);

    hipFree(input);
    free(output_cpu);
    hipFree(output_gpu);
    hipFree(output_gpu_double_shared);
    hipFree(partialSums1);
    hipFree(partialSums2);
    
    return 0;
}
