#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define N (1<<24)
#define THREADS 1024

__global__ void fp64(const double* input, int8_t* output, float scale)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) 
    {
        double x = input[idx];
        int q = __float2int_rn((float)(x / scale));
        output[idx] = max(-128, min(127, q));
    }
}

__global__ void fp32(const float* input, int8_t* output, float scale)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) 
    {
        float x = input[idx];
        int q = __float2int_rn(x / scale);
        output[idx] = max(-128, min(127, q));
    }
}

__global__ void fp16(const half* input, int8_t* output, float scale)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) 
    {
        float x = __half2float(input[idx]);
        int q = __float2int_rn(x / scale);
        output[idx] = max(-128, min(127, q));
    }
}

void init(double* d64, float* d32, half* d16)
{
    for (int i = 0; i < N; ++i) 
    {
        float val = sinf(i * 0.001f) * 10.0f;
        d64[i] = (double)val;
        d32[i] = val;
        d16[i] = __float2half(val);
    }
}

int main()
{
    double* d_fp64;
    float*  d_fp32;
    half*   d_fp16;
    int8_t *d_out64, *d_out32, *d_out16;
    hipMallocManaged(&d_fp64, N * sizeof(double));
    hipMallocManaged(&d_fp32, N * sizeof(float));
    hipMallocManaged(&d_fp16, N * sizeof(half));
    hipMallocManaged(&d_out64, N * sizeof(int8_t));
    hipMallocManaged(&d_out32, N * sizeof(int8_t));
    hipMallocManaged(&d_out16, N * sizeof(int8_t));

    init(d_fp64, d_fp32, d_fp16);

    dim3 threads(THREADS);
    dim3 blocks((N + THREADS - 1) / THREADS);
    float scale = 0.1f;

    hipEvent_t start, stop;
    float time_fp64 = 0, time_fp32 = 0, time_fp16 = 0;

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    fp64<<<blocks, threads>>>(d_fp64, d_out64, scale);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_fp64, start, stop);
    hipEventDestroy(start); hipEventDestroy(stop);

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    fp32<<<blocks, threads>>>(d_fp32, d_out32, scale);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_fp32, start, stop);
    hipEventDestroy(start); hipEventDestroy(stop);

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    fp16<<<blocks, threads>>>(d_fp16, d_out16, scale);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_fp16, start, stop);
    hipEventDestroy(start); hipEventDestroy(stop);

    printf("FP64 -> int8: %.3f ms\n", time_fp64);
    printf("FP32 -> int8: %.3f ms\n", time_fp32);
    printf("FP16 -> int8: %.3f ms\n", time_fp16);

    hipFree(d_fp64); 
    hipFree(d_fp32); 
    hipFree(d_fp16);
    hipFree(d_out64); 
    hipFree(d_out32); 
    hipFree(d_out16);

    return 0;
}
