#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10 * 1000000

#define THREADS 1024
#define SAMPLES_PER_THREAD (N / THREADS + 1)
#define TOTAL_THREADS ((N + SAMPLES_PER_THREAD - 1) / SAMPLES_PER_THREAD)
#define BLOCKS ((TOTAL_THREADS + THREADS - 1) / THREADS)

__global__ void monte_carlo_warp(int *res, int samples_per_thread)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = threadIdx.x % warpSize;

    hiprandState state;
    hiprand_init(idx, 0, 0, &state);

    int count = 0;
    for (int i = 0; i < samples_per_thread; ++i)
    {
        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0f)
        {
            count++;
        }
    }

    for (int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        count += __shfl_down_sync(0xFFFFFFFF, count, offset);
    }

    if (lane == 0)
    {
        atomicAdd(res, count);
    }
}


__global__ void monte_carlo_thread(int *res, int samples_per_thread) 
{
    __shared__ int shared_counts[THREADS];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(idx, 0, 0, &state);

    int local_count = 0;
    for (int i = 0; i < samples_per_thread; ++i) 
    {
        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0f) local_count++;
    }

    shared_counts[threadIdx.x] = local_count;
    __syncthreads();

    if (threadIdx.x == 0) 
    {
        int block_sum = 0;
        for (int i = 0; i < blockDim.x; i++) 
        {
            block_sum += shared_counts[i];
        }
        atomicAdd(res, block_sum);
    }
}

int main() 
{
    int *res_warp, *res_thread;
    hipMallocManaged(&res_warp, sizeof(int));
    hipMallocManaged(&res_thread, sizeof(int));

    hipEvent_t start, stop;
    float time_warp, time_thread;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemset(res_warp, 0, sizeof(int));
    hipEventRecord(start);
    monte_carlo_warp<<<BLOCKS, THREADS>>>(res_warp, SAMPLES_PER_THREAD);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_warp, start, stop);
    float pi_warp = (4.0f * (*res_warp)) / N;

    hipMemset(res_thread, 0, sizeof(int));
    hipEventRecord(start);
    monte_carlo_thread<<<BLOCKS, THREADS>>>(res_thread, SAMPLES_PER_THREAD);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_thread, start, stop);
    float pi_thread = (4.0f * (*res_thread)) / N;

    hipFree(res_thread);
    hipFree(res_warp);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Warp Sync:- Pi %f, Time %f ms\n", pi_warp, time_warp);
    printf("Thread Sync:- Pi %f, Time %f ms\n", pi_thread, time_thread);

    return 0;
}
