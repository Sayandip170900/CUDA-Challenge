#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void multi_gpu(int idx) 
{
    printf("Hello from GPU %d (block %d, thread %d)\n", idx, blockIdx.x, threadIdx.x);
}

int main() {
    int n;
    hipGetDeviceCount(&n);
    printf("Total GPUs: %d\n", n);

    for (int i = 0; i < n; ++i) 
    {
        hipSetDevice(i);

        printf("Launching kernel on GPU %d\n", i);

        multi_gpu<<<2, 2>>>(i);

        hipDeviceSynchronize();
    }

    return 0;
}
