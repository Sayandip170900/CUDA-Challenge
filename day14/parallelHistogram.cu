
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define N (1<<15)
#define THREADS 256
#define BLOCKS ((THREADS + N - 1) / THREADS)

#define BINS 5
#define TOTAL_BINS (BLOCKS * BINS)

void init(unsigned int *data)
{
    srand(time(NULL));
    
    for (unsigned int i = 0; i < N; i++)
    {
        data[i] = (rand() % 100) + 1;
    }
}

void histCPU(unsigned int *data, unsigned int *hist)
{
    for(unsigned int i = 0; i < N; i++)
    {
        if(data[i] > 0 && data[i] <= 100)
        {
            hist[(data[i] - 1)/20] += 1;
        }
    }
}

__global__ void histGPU(unsigned int *data, unsigned int *hist)
{
    unsigned int t_i = blockIdx.x * blockDim.x + threadIdx.x;
    if(t_i < N)
    {
        if(data[t_i] > 0 && data[t_i] <= 100)
        {
            atomicAdd(&hist[(data[t_i] - 1)/20], 1);
        }
    }
}

__global__ void histPvtGPU(unsigned int *data, unsigned int *hist)
{
    unsigned int t_i = blockIdx.x * blockDim.x + threadIdx.x;
    if(t_i < N)
    {
        if(data[t_i] > 0 && data[t_i] <= 100)
        {
            atomicAdd(&hist[(blockIdx.x * BINS + (data[t_i] - 1)/20)], 1);
        }
    }

    if(blockIdx.x > 0)
    {
        __syncthreads();
        for(unsigned bin = threadIdx.x; bin < BINS; bin += blockDim.x)
        {
            unsigned int val = hist[blockIdx.x * BINS + bin];
            if(val > 0)
            {
                atomicAdd(&hist[bin], val);
            }
        }
    }
}

__global__ void histPvtSharedGPU(unsigned int *data, unsigned int *hist)
{
    unsigned int t_i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ unsigned int hist_s[BINS];
    for(unsigned bin = threadIdx.x; bin < BINS; bin += blockDim.x)
    {
        hist_s[bin] = 0u;
    }
    __syncthreads();

    if(t_i < N)
    {
        if(data[t_i] > 0 && data[t_i] <= 100)
        {
            atomicAdd(&hist_s[(data[t_i] - 1)/20], 1);
        }
    }
    __syncthreads();

    for(unsigned bin = threadIdx.x; bin < BINS; bin += blockDim.x)
    {   
        if(hist_s[bin] > 0)
        {
            atomicAdd(&hist[bin], hist_s[bin]);
        }
    }
}

__global__ void verify(unsigned int *hist_cpu, unsigned int *hist_gpu, unsigned int *hist_gpu_pvt, unsigned int *hist_gpu_pvt_shared, unsigned int *errors)
{
    unsigned int t_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (t_i < BINS)
    {
        if (hist_cpu[t_i] != hist_gpu[t_i] || hist_gpu[t_i] != hist_gpu_pvt[t_i] || hist_gpu_pvt[t_i] != hist_gpu_pvt_shared[t_i])
        {
            atomicAdd(errors, 1);
        }
    }
}

int main()
{
    unsigned int *data, *hist_cpu, *hist_gpu, *hist_gpu_pvt, *hist_gpu_pvt_shared;

    size_t size = N * sizeof(unsigned int);

    hipMallocManaged(&data, size);
    hipMallocManaged(&hist_cpu, BINS * sizeof(unsigned int));
    hipMallocManaged(&hist_gpu, BINS * sizeof(unsigned int));
    hipMallocManaged(&hist_gpu_pvt, TOTAL_BINS * sizeof(unsigned int));
    hipMallocManaged(&hist_gpu_pvt_shared, BINS * sizeof(unsigned int));

    hipMemset(hist_cpu, 0, BINS * sizeof(unsigned int));
    hipMemset(hist_gpu, 0, BINS * sizeof(unsigned int));
    hipMemset(hist_gpu_pvt, 0, TOTAL_BINS * sizeof(unsigned int));
    hipMemset(hist_gpu_pvt_shared, 0, BINS * sizeof(unsigned int));

    init(data);

    float cpu_time, gpu_time, gpu_time_pvt, gpu_time_pvt_shared;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histGPU<<<BLOCKS, THREADS>>>(data, hist_gpu);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    histPvtGPU<<<BLOCKS, THREADS>>>(data, hist_gpu_pvt);
    hipDeviceSynchronize();
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&gpu_time_pvt, start2, stop2);

    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    hipEventRecord(start3);
    histPvtSharedGPU<<<BLOCKS, THREADS>>>(data, hist_gpu_pvt_shared);
    hipDeviceSynchronize();
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    hipEventElapsedTime(&gpu_time_pvt_shared, start3, stop3);

    clock_t cpu_start = clock();
    histCPU(data, hist_cpu);
    cpu_time = ((double)(clock() - cpu_start) / CLOCKS_PER_SEC) * 1000;

    unsigned int *errors;
    hipMallocManaged(&errors, sizeof(unsigned int));
    verify<<<1, 5>>>(hist_cpu, hist_gpu, hist_gpu_pvt, hist_gpu_pvt_shared, errors);
    hipDeviceSynchronize();

    printf("\nNumber of mismatches: %d\n", *errors);
    printf("\nCPU execution time: %.4f ms\n", cpu_time);
    printf("GPU execution time: %.4f ms\n", gpu_time);
    printf("GPU execution time with privatization of input: %.4f ms\n", gpu_time_pvt);
    printf("GPU execution time with privatization of input in shared memory: %.4f ms\n", gpu_time_pvt_shared);

    printf("\nHistogram:\n");
    for (int i = 0; i < BINS; i++)
    {
        printf("Bin %d has %u numbers\n", i, hist_cpu[i]);
    }

    hipFree(data);
    hipFree(hist_cpu);
    hipFree(hist_gpu);
    hipFree(hist_gpu_pvt);
    hipFree(hist_gpu_pvt_shared);
    hipFree(errors);

    return 0;
}