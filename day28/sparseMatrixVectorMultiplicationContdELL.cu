#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

#define N (1<<12)
#define THREADS 1024
#define BLOCKS ((N * N + THREADS - 1) / THREADS)
#define SPARSITY 0.1f

__global__ void init(int *sparse_m, unsigned long seed) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = N * N;
    if (tid < totalElements) 
    {
        hiprandState state;
        hiprand_init(seed, tid, 0, &state);
        float rand_val = hiprand_uniform(&state);
        sparse_m[tid] = (rand_val <= SPARSITY) ? (hiprand(&state) % 100 + 1) : 0;
    }
}

__global__ void initVect(int *vect, unsigned long seed) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) 
    {
        hiprandState state;
        hiprand_init(seed, tid, 0, &state);
        float rand_val = hiprand_uniform(&state);
        vect[tid] = hiprand(&state) % 100 + 1;
    }
}

void COO(int *sparse_m, int **cooRow, int **cooCol, int **cooVal, int64_t *non_zero) 
{
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseDnMatDescr_t matrix;
    hipsparseCreateDnMat(&matrix, N, N, N, sparse_m, HIP_R_32I, HIPSPARSE_ORDER_ROW);

    *non_zero = 0;
    for (int i = 0; i < N * N; i++) 
    {
        if (sparse_m[i] != 0)
            (*non_zero)++;
    }

    hipMallocManaged(cooRow, (*non_zero) * sizeof(int));
    hipMallocManaged(cooCol, (*non_zero) * sizeof(int));
    hipMallocManaged(cooVal, (*non_zero) * sizeof(int));

    hipsparseSpMatDescr_t matCOO;
    hipsparseCreateCoo(&matCOO, N, N, *non_zero, *cooRow, *cooCol, *cooVal,
                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I);

    size_t bufferSize = 0;
    void *dBuffer = NULL;
    hipsparseDenseToSparseAlg_t alg = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
    
    hipsparseDenseToSparse_bufferSize(handle, matrix, matCOO, alg, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseDenseToSparse_analysis(handle, matrix, matCOO, alg, &bufferSize);
    hipsparseDenseToSparse_convert(handle, matrix, matCOO, alg, dBuffer);

    hipFree(dBuffer);
    hipsparseDestroyDnMat(matrix);
    hipsparseDestroySpMat(matCOO);
    hipsparseDestroy(handle);
}

void CSR(int *sparse_m, int **csrRowPtr, int **csrCol, int **csrVal, int64_t *non_zero) 
{
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseDnMatDescr_t matrix;
    hipsparseCreateDnMat(&matrix, N, N, N, sparse_m, HIP_R_32I, HIPSPARSE_ORDER_ROW);

    *non_zero = 0;
    for (int i = 0; i < N * N; i++) 
    {
        if (sparse_m[i] != 0)
            (*non_zero)++;
    }

    hipMallocManaged(csrRowPtr, (N + 1) * sizeof(int));
    hipMallocManaged(csrCol, (*non_zero) * sizeof(int));
    hipMallocManaged(csrVal, (*non_zero) * sizeof(int));

    hipsparseSpMatDescr_t matCSR;
    hipsparseCreateCsr(&matCSR, N, N, *non_zero, *csrRowPtr, *csrCol, *csrVal,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I);

    size_t bufferSize = 0;
    void *dBuffer = NULL;
    hipsparseDenseToSparseAlg_t alg = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
    
    hipsparseDenseToSparse_bufferSize(handle, matrix, matCSR, alg, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseDenseToSparse_analysis(handle, matrix, matCSR, alg, &bufferSize);
    hipsparseDenseToSparse_convert(handle, matrix, matCSR, alg, dBuffer);

    hipFree(dBuffer);
    hipsparseDestroyDnMat(matrix);
    hipsparseDestroySpMat(matCSR);
    hipsparseDestroy(handle);
}

void ELL(int *sparse_m, int **ellCol, int **ellVal, int *max_nnz) {
    int *row_nnz = (int*) malloc(N * sizeof(int));
    *max_nnz = 0;
    for (int i = 0; i < N; i++) 
    {
        int count = 0;
        for (int j = 0; j < N; j++) 
        {
            if (sparse_m[i * N + j] != 0)
                count++;
        }
        row_nnz[i] = count;
        if (count > *max_nnz)
            *max_nnz = count;
    }

    hipMallocManaged(ellVal, N * (*max_nnz) * sizeof(int));
    hipMallocManaged(ellCol, N * (*max_nnz) * sizeof(int));

    for (int i = 0; i < N * (*max_nnz); i++) 
    {
        (*ellVal)[i] = 0;
        (*ellCol)[i] = -1;
    }

    for (int i = 0; i < N; i++)
    {
        int count = 0;
        for (int j = 0; j < N; j++) 
        {
            int val = sparse_m[i * N + j];
            if (val != 0) 
            {
                int index = i * (*max_nnz) + count;
                (*ellVal)[index] = val;
                (*ellCol)[index] = j;
                count++;
            }
        }
    }
    free(row_nnz);
}

struct row_data 
{
    int index;
    int nnz;
};

int cmp_row_data(const void *a, const void *b) 
{
    const struct row_data *ra = (const struct row_data*) a;
    const struct row_data *rb = (const struct row_data*) b;
    return rb->nnz - ra->nnz;
}

void JDS(int *sparse_m, int **jdsRow, int **jdsCol, int **jdsVal, int **jdsPerm, int64_t *non_zero, int *max_nnz) 
{
    int *row_nnz = (int*) malloc(N * sizeof(int));
    int *perm = (int*) malloc(N * sizeof(int));
    *non_zero = 0;

    for (int i = 0; i < N; i++) 
    {
        int count = 0;
        for (int j = 0; j < N; j++) 
        {
            if (sparse_m[i * N + j] != 0)
                count++;
        }
        row_nnz[i] = count;
        perm[i] = i;
        *non_zero += count;
    }

    struct row_data *rows = (struct row_data*) malloc(N * sizeof(struct row_data));
    for (int i = 0; i < N; i++) 
    {
        rows[i].index = i;
        rows[i].nnz = row_nnz[i];
    }
    qsort(rows, N, sizeof(struct row_data), cmp_row_data);
    for (int i = 0; i < N; i++) 
    {
        perm[i] = rows[i].index;
    }

    int jds_max = (N > 0 ? rows[0].nnz : 0);
    *max_nnz = jds_max;

    hipMallocManaged(jdsRow, (jds_max + 1) * sizeof(int));
    hipMallocManaged(jdsCol, (*non_zero) * sizeof(int));
    hipMallocManaged(jdsVal, (*non_zero) * sizeof(int));
    hipMallocManaged(jdsPerm, N * sizeof(int));

    for (int i = 0; i < N; i++) 
    {
        (*jdsPerm)[i] = perm[i];
    }

    int idx = 0;
    (*jdsRow)[0] = 0;
    for (int d = 0; d < jds_max; d++) 
    {
        int count = 0;
        for (int i = 0; i < N; i++) 
        {
            if (row_nnz[perm[i]] > d)
                count++;
            else
                break;
        }
        idx += count;
        (*jdsRow)[d+1] = idx;
    }

    int pos = 0;
    for (int d = 0; d < jds_max; d++) 
    {
        for (int i = 0; i < N; i++) 
        {
            int row = perm[i];
            if (row_nnz[row] > d) 
            {
                int count = 0;
                for (int j = 0; j < N; j++) 
                {
                    if (sparse_m[row * N + j] != 0) 
                    {
                        if (count == d) 
                        {
                            (*jdsVal)[pos] = sparse_m[row * N + j];
                            (*jdsCol)[pos] = j;
                            pos++;
                            break;
                        }
                        count++;
                    }
                }
            } 
            else 
            {
                break;
            }
        }
    }
    free(row_nnz);
    free(perm);
    free(rows);
}

__global__ void spMV_COO(int *cooRow, int *cooCol, int *cooVal, int *in_vect, int *out_vect, int64_t non_zero)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < non_zero)
    {
        int row = cooRow[tid];
        int col = cooCol[tid];
        int val = cooVal[tid];
        atomicAdd(&out_vect[row], val * in_vect[col]);
    }
}

__global__ void spMV_CSR(int *csrRowPtr, int *csrCol, int *csrVal, int *in_vect, int *out_vect, int num_rows)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_rows)
    {
        int sum = 0;
        for (int i = csrRowPtr[tid]; i < csrRowPtr[tid + 1]; i++)
        {
            int col = csrCol[i];
            int val = csrVal[i];
            sum += in_vect[col] * val;
        }
        out_vect[tid] = sum;
    }
}

__global__ void spMV_ELL(int *ellCol, int *ellVal, int *in_vect, int *out_vect, int ell_max_nnz)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N)
    {
        int sum = 0;
        for (int d = 0; d < ell_max_nnz; d++)
        {
            int idx = row * ell_max_nnz + d;
            int col = ellCol[idx];
            int val = ellVal[idx];
            if (col != -1)
            {
                sum += val * in_vect[col];
            }
        }
        out_vect[row] = sum;
    }
}

void spMV_CPU(int *sparse_m, int *in_vect, int *out_vect)
{
    for (int row = 0; row < N; row++) 
    {
        int sum = 0;
        for (int col = 0; col < N; col++) 
        {
            sum += sparse_m[row * N + col] * in_vect[col];
        }
        out_vect[row] = sum;
    }
}

int main() 
{
    int *sparse_m, *in_vect;
    hipMallocManaged(&sparse_m, N * N * sizeof(int));
    hipMallocManaged(&in_vect, N * sizeof(int));

    int *out_vect_coo, *out_vect_csr, *out_vect_ell, *out_vect_cpu;
    hipMallocManaged(&out_vect_coo, N * sizeof(int));
    hipMallocManaged(&out_vect_csr, N * sizeof(int));
    hipMallocManaged(&out_vect_ell, N * sizeof(int));
    hipMallocManaged(&out_vect_cpu, N * sizeof(int));
    
    hipMemset(out_vect_coo, 0, N * sizeof(int));
    hipMemset(out_vect_csr, 0, N * sizeof(int));
    hipMemset(out_vect_cpu, 0, N * sizeof(int));

    init<<<BLOCKS, THREADS>>>(sparse_m, time(NULL));
    hipDeviceSynchronize();
    initVect<<<((N + THREADS - 1)/THREADS), THREADS>>>(in_vect, time(NULL));
    hipDeviceSynchronize();

    clock_t cpu_start = clock();
    spMV_CPU(sparse_m, in_vect, out_vect_cpu);
    float cpu_time = ((double)(clock() - cpu_start) / CLOCKS_PER_SEC) * 1000;

    int *cooRow, *cooCol, *cooVal;
    int64_t non_zero_coo = 0;
    clock_t start_coo_creation = clock();
    COO(sparse_m, &cooRow, &cooCol, &cooVal, &non_zero_coo);
    clock_t end_coo_creation = clock();
    double coo_creation_time = ((double)(end_coo_creation - start_coo_creation)) / CLOCKS_PER_SEC * 1000.0;

    hipEvent_t start, stop;
    float coo_operation_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    spMV_COO<<<((non_zero_coo + THREADS - 1) / THREADS), THREADS>>>(cooRow, cooCol, cooVal, in_vect, out_vect_coo, non_zero_coo);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&coo_operation_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    int *csrRowPtr, *csrCol, *csrVal;
    int64_t non_zero_csr = 0;
    clock_t start_csr_creation = clock();
    CSR(sparse_m, &csrRowPtr, &csrCol, &csrVal, &non_zero_csr);
    clock_t end_csr_creation = clock();
    double csr_creation_time = ((double)(end_csr_creation - start_csr_creation)) / CLOCKS_PER_SEC * 1000.0;

    int num_rows_csr = sizeof(csrRowPtr) / sizeof(csrRowPtr[0]);
    hipEvent_t start2, stop2;
    float csr_operation_time;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    spMV_CSR<<<((num_rows_csr + THREADS - 1) / THREADS), THREADS>>>(csrRowPtr, csrCol, csrVal, in_vect, out_vect_csr, num_rows_csr);
    hipDeviceSynchronize();
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&csr_operation_time, start2, stop2);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    int *ellCol, *ellVal, ell_max_nnz;
    clock_t start_ell_creation = clock();
    ELL(sparse_m, &ellCol, &ellVal, &ell_max_nnz);
    clock_t end_ell_creation = clock();
    double ell_creation_time = ((double)(end_ell_creation - start_ell_creation)) / CLOCKS_PER_SEC * 1000.0;

    hipEvent_t start3, stop3;
    float ell_operation_time;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    hipEventRecord(start3);
    spMV_ELL<<<((N + THREADS - 1) / THREADS), THREADS>>>(ellCol, ellVal, in_vect, out_vect_ell, ell_max_nnz);
    hipDeviceSynchronize();
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    hipEventElapsedTime(&ell_operation_time, start3, stop3);
    hipEventDestroy(start3);
    hipEventDestroy(stop3);
    
    int *jdsRow, *jdsCol, *jdsVal, *jdsPerm;
    int64_t non_zero_jds = 0;
    int jds_max_nnz;
    clock_t start_jds_creation = clock();
    JDS(sparse_m, &jdsRow, &jdsCol, &jdsVal, &jdsPerm, &non_zero_jds, &jds_max_nnz);
    clock_t end_jds_creation = clock();
    double jds_creation_time = ((double)(end_jds_creation - start_jds_creation)) / CLOCKS_PER_SEC * 1000.0;
    
    double coo_total_time = coo_creation_time + coo_operation_time;
    double csr_total_time = csr_creation_time + csr_operation_time;
    double ell_total_time = ell_creation_time + ell_operation_time;

    printf("Dense Matrix Storage Size: %.2lf MB\n", (N * N * sizeof(int) / (1024.0 * 1024)));
    printf("Sparse Matrix Storage Size (COO): %.2lf MB\n", ((non_zero_coo * 3 * sizeof(int)) / (1024.0 * 1024)));
    printf("Sparse Matrix Storage Size (CSR): %.2lf MB\n", (((N + 1) * sizeof(int) + non_zero_csr * 2 * sizeof(int)) / (1024.0 * 1024)));
    printf("Sparse Matrix Storage Size (ELL): %.2lf MB\n", ((N * ell_max_nnz * 2 * sizeof(int)) / (1024.0 * 1024)));
    printf("Sparse Matrix Storage Size (JDS): %.2lf MB\n\n", ((((jds_max_nnz + 1) * sizeof(int)) + non_zero_jds * 2 * sizeof(int) + N * sizeof(int)) / (1024.0 * 1024)));
    
    printf("COO Creation Time: %f ms\n", coo_creation_time);
    printf("COO Operation Time: %f ms\n", coo_operation_time);
    printf("COO Total Time: %f ms\n\n", coo_total_time);
    
    printf("CSR Creation Time: %f ms\n", csr_creation_time);
    printf("CSR Operation Time: %f ms\n", csr_operation_time);
    printf("CSR Total Time: %f ms\n\n", csr_total_time);

    printf("ELL Creation Time: %f ms\n", ell_creation_time);
    printf("ELL Operation Time: %f ms\n", ell_operation_time);
    printf("ELL Total Time: %f ms\n\n", ell_total_time);
    
    printf("CPU spMV Time: %f ms\n\n", cpu_time);
    
    hipFree(sparse_m);
    hipFree(in_vect);
    hipFree(out_vect_coo);
    hipFree(out_vect_csr);
    hipFree(out_vect_cpu);
    hipFree(cooRow);
    hipFree(cooCol);
    hipFree(cooVal);
    hipFree(csrRowPtr);
    hipFree(csrCol);
    hipFree(csrVal);
    hipFree(ellCol);
    hipFree(ellVal);
    hipFree(jdsRow);
    hipFree(jdsCol);
    hipFree(jdsVal);
    hipFree(jdsPerm);
    
    return 0;
}
