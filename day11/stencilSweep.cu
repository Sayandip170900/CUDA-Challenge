
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

__constant__ float val[7];

__global__ void stencilGPU(float *input, float *output, float *val, int N)
{
    int depth = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(depth >=1 && depth < N - 1 && row >=1 && row < N - 1 && col >=1 && col < N - 1)
    {   
        output[depth * N * N + row * N + col] = val[0] * input[depth * N * N + row * N + col] +
                                                val[1] * input[depth * N * N + row * N + col - 1] +
                                                val[2] * input[depth * N * N + row * N + col + 1] +
                                                val[3] * input[depth * N * N + (row - 1) * N + col] +
                                                val[4] * input[depth * N * N + (row + 1) * N + col] +
                                                val[5] * input[(depth - 1) * N * N + row * N + col] +
                                                val[6] * input[(depth + 1) * N * N + row * N + col];
    }    
}

__global__ void init(float *input, int N, unsigned int seed)
{
    int depth = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(depth < N && row < N && col < N)
    {
        hiprandState state;
        hiprand_init(seed, depth * N * N + row * N + col, 0, &state);
        input[depth * N * N + row * N + col] = hiprand_uniform(&state);
    }
}

void stencilCPU(float *input, float *output, float *val, int N)
{
    for (int depth = 1; depth < N - 1; depth++)
    {
        for (int row = 1; row < N - 1; row++)
        {
            for (int col = 1; col < N - 1; col++)
            {
                output[depth * N * N + row * N + col] = val[0] * input[depth * N * N + row * N + col] +
                                                        val[1] * input[depth * N * N + row * N + col - 1] +
                                                        val[2] * input[depth * N * N + row * N + col + 1] +
                                                        val[3] * input[depth * N * N + (row - 1) * N + col] +
                                                        val[4] * input[depth * N * N + (row + 1) * N + col] +
                                                        val[5] * input[(depth - 1) * N * N + row * N + col] +
                                                        val[6] * input[(depth + 1) * N * N + row * N + col];
            }
        }
    }
}

__global__ void verify(float *output_cpu, float *output_gpu, int N, int *errors)
{
    int depth = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (depth >= 1 && depth < N - 1 && row >= 1 && row < N - 1 && col >= 1 && col < N - 1)
    {
        if (fabs(output_cpu[depth * N * N + row * N + col] - output_gpu[depth * N * N + row * N + col]) > 1e-4)
        {
            atomicAdd(errors, 1);
        }
    }
}

int main()
{
    int N = 128;
    float *input, *output_gpu, *output_cpu;

    size_t size = N * N * N * sizeof(float);
    hipMallocManaged(&input, size);
    hipMallocManaged(&output_gpu, size);
    hipMallocManaged(&output_cpu, size);

    hipMemset(output_gpu, 0, size);
    hipMemset(output_cpu, 0, size);

    dim3 threads(8, 8, 8);
    dim3 blocks((N + threads.x - 1)/threads.x, (N + threads.y - 1)/threads.y, (N + threads.z - 1)/threads.z);

    init<<<blocks, threads>>>(input, N, time(NULL));
    hipDeviceSynchronize();

    float *val;
    hipMallocManaged(&val, 7 * sizeof(float));
    val[0] = 1.0f;
    val[1] = 2.0f;
    val[2] = 3.0f;
    val[3] = 4.0f;
    val[4] = 5.0f;
    val[5] = 6.0f;
    val[6] = 7.0f;

    float gpu_time, cpu_time;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    stencilGPU<<<blocks, threads>>>(input, output_gpu, val, N);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    clock_t cpu_start = clock();
    stencilCPU(input, output_cpu, val, N);
    cpu_time = ((double)(clock() - cpu_start) / CLOCKS_PER_SEC) * 1000;

    int *errors;
    hipMallocManaged(&errors, sizeof(int));
    verify<<<blocks, threads>>>(output_cpu, output_gpu, N, errors);
    hipDeviceSynchronize();

    printf("Number of mismatches: %d\n", *errors);
    printf("GPU time: %f ms\n", gpu_time);
    printf("CPU time: %f ms\n", cpu_time);

    hipFree(input);
    hipFree(output_gpu);
    hipFree(output_cpu);
    hipFree(val);
    hipFree(errors);

    return 0;
}